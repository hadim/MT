#include "hip/hip_runtime.h"
/*
 * compute_cuda.cu
 *
 *  Created on: 04.06.2012
 *      Author: zhmurov

   Modified on: 09.01.2015
   		Author: klyshko
 */
#include "compute_cuda.cuh"

__device__ __constant__ Parameters c_par;
__device__ __constant__ Topology c_top;
__device__ __constant__ Tea c_tea;

__device__ real dmorse(real D, real a, real x){
    return (2*a*D*(1-exp(-a*x))*exp(-a*x));
}

__device__ real morse_en(real D, real a, real x){
    return D * (1 - exp(-a*x)) * (1 - exp(-a*x)) - D;
}

__device__ real dbarr(real a, real r, real w, real x){
    return ( - a*exp(-(x-r)*(x-r)/(2*w*w)) * (x-r)/(w*w));
}

__device__ real barr(real a, real r, real w, real x){
    return a*exp(-(x-r)*(x-r)/(2*w*w));
}

__global__ void compute_kernel(const Coord* d_r, Coord* d_f){
	const int p = blockIdx.x*blockDim.x + threadIdx.x;
	const int ind = p%c_par.Ntot;
	const int traj = p/c_par.Ntot;
	real cos_fii, cos_fij, sin_fii, sin_fij, 
		  cos_psii, cos_psij, sin_psii, sin_psij,
		  cos_thetai, cos_thetaj, sin_thetai, sin_thetaj;
	real xi, xj, yi, yj, zi, zj;
	real dUdr, dr, gradx, grady, gradz, gradfi, gradpsi, gradtheta;
	int j;
	real psiji, thetaji, fiji, psiij, thetaij, fiij;
	Coord ri, rj, fi = (Coord){0.0,0.0,0.0,0.0,0.0,0.0};

	real xp1 = xp1_def;
	real yp1 = yp1_def;
	real zp1 = zp1_def;
	real xp2 = xp2_def;
	real yp2 = yp2_def;
	real zp2 = zp2_def;
	real R_MON = r_mon;

	if(ind < c_par.Ntot && traj < c_par.Ntr){
		
		if (!c_top.extra[ind + traj * c_par.Ntot]){

			
			ri = d_r[p];
			xi = ri.x;
			yi = ri.y;
			zi = ri.z; 
			cos_fii = cosf(ri.fi); 
			sin_fii = sinf(ri.fi);
			cos_psii = cosf(ri.psi);
			sin_psii = sinf(ri.psi);
			cos_thetai = cosf(ri.theta);
			sin_thetai = sinf(ri.theta);
			
			// harmonic
			for(int k = 0; k < c_top.harmonicCount[ind]; k++){
				j = c_top.harmonic[c_top.maxHarmonicPerMonomer*ind+k];
				if(j < 0){
					R_MON = r_mon;
					j *= -1;
				}
				else{
					R_MON = -r_mon;
				}
				rj = d_r[j + traj*c_par.Ntot];
				cos_fij = cosf(rj.fi);
				sin_fij = sinf(rj.fi);
				cos_psij = cosf(rj.psi);
				sin_psij = sinf(rj.psi);
				cos_thetaj = cosf(rj.theta);
				sin_thetaj = sinf(rj.theta);
				xj = rj.x;
				yj = rj.y;
				zj = rj.z;
				dr = sqrtf(pow(-zi + zj -
			 		R_MON * cos_fii * cos_thetai - R_MON * cos_fij * cos_thetaj,2) +
		   			pow(-xi + xj -
			 		R_MON * (sin_fii * sin_psii + cos_fii * cos_psii * sin_thetai) -
					R_MON * (sin_fij * sin_psij + cos_fij * cos_psij * sin_thetaj),2) +
					pow(-yi + yj -
					R_MON * (-cos_psii * sin_fii + cos_fii * sin_psii * sin_thetai) -
					R_MON * (-cos_psij * sin_fij + cos_fij * sin_psij * sin_thetaj),2));

				gradx = -((-xi + xj - R_MON* (sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
						R_MON* (sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj)));


				grady = -((-yi + yj -
						R_MON* (-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
						R_MON* (-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj)));


				gradz = -((-zi + zj - R_MON* cos_fii*cos_thetai - R_MON* cos_fij*cos_thetaj));


				gradtheta = ( R_MON*(-zi + zj - R_MON* cos_fii*cos_thetai - R_MON* cos_fij*cos_thetaj)*cos_fii*sin_thetai -
							R_MON* (-xi + xj -
							R_MON* (sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
							R_MON* (sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj))*cos_fii*cos_psii*cos_thetai -
							R_MON* (-yi + yj - R_MON* (-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
							R_MON* (-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj))*cos_fii*cos_thetai*sin_psii);


				gradpsi = (-R_MON*(-xi + xj - R_MON*(sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
							R_MON*(sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj))*(cos_psii*sin_fii - cos_fii*sin_psii*sin_thetai) -
							R_MON*(-yi + yj - R_MON*(-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
							R_MON*(-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj))*(sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai));


				gradfi = ( R_MON* (-zi + zj -
						R_MON* cos_fii*cos_thetai - R_MON* cos_fij*cos_thetaj)*cos_thetai*sin_fii -
						R_MON* (-xi + xj -
						R_MON* (sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
						R_MON* (sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj))*(cos_fii*sin_psii - cos_psii*sin_fii*sin_thetai) -
						R_MON* (-yi + yj - R_MON* (-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
						R_MON* (-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj))*(-cos_fii*cos_psii - sin_fii*sin_psii*sin_thetai));


				fi.x     += -c_par.C * gradx;
				fi.y     += -c_par.C * grady;
				fi.z     += -c_par.C * gradz;
				fi.fi    += -c_par.C * gradfi;
				fi.psi   += -c_par.C * gradpsi;	
				fi.theta += -c_par.C * gradtheta;
			
			
	            if(dr < ANGLE_CUTOFF )
	            {
	            	/*psiji = rj.psi - ri.psi - 2 * M_PI * (int)((rj.psi - ri.psi)/(2 * M_PI));
	                psiij = - psiji;
	            	thetaji = rj.theta - ri.theta - 2 * M_PI * (int)((rj.theta - ri.theta)/(2 * M_PI));
	            	thetaij = - thetaji;
	              	fiji = rj.fi - ri.fi - 2 * M_PI * (int)((rj.fi - ri.fi)/(2 * M_PI));
	                fiij = - fiji;
	*/				psiji = rj.psi - ri.psi;
	                psiij = - psiji;
	            	thetaji = rj.theta - ri.theta;
	            	thetaij = - thetaji;
	              	fiji = rj.fi - ri.fi;
	                fiij = - fiji;
	                
	                if(R_MON > 0){

	                    fi.psi   += c_par.B_psi		*	sinf(psiji 		- c_par.psi_0);
	                    fi.fi	 += c_par.B_fi		*	sinf(fiji 		- c_par.fi_0	);

	                    if (c_top.gtp[p] == 1){
	                    	fi.theta += c_par.B_theta	*	sinf(thetaji 	- c_par.theta0_gtp	);
	                    } else{
	                    	fi.theta += c_par.B_theta	*	sinf(thetaji 	- c_par.theta0_gdp	);
	                    }	                    
	                }
	                else{

	                    fi.psi   -= c_par.B_psi		*	sinf(psiij	- 	c_par.psi_0	);
	                    fi.fi	 -= c_par.B_fi		*	sinf(fiij 		- c_par.fi_0	);

	                    if (c_top.gtp[p] == 1){
	                    	fi.theta -= c_par.B_theta	*	sinf(thetaij 	- c_par.theta0_gtp	);
	                    } else{
	                    	fi.theta -= c_par.B_theta	*	sinf(thetaij 	- c_par.theta0_gdp	);
	                    }
	                }
	                
	              	
	            }
	            
			}
			
			
			
			
#if defined(MORSE)
			
			for(int k = 0; k < c_top.longitudinalCount[ind + c_par.Ntot * traj]; k++){
				j = c_top.longitudinal[c_top.maxLongitudinalPerMonomer * c_par.Ntot * traj + ind * c_top.maxLongitudinalPerMonomer + k];
				if(j < 0){
					R_MON = r_mon;
					j = abs(j);
				}
				else {
					R_MON = -r_mon;
				}
				rj = d_r[j + traj*c_par.Ntot];
				cos_fij = cosf(rj.fi);
				sin_fij = sinf(rj.fi);
				cos_psij = cosf(rj.psi);
				sin_psij = sinf(rj.psi);
				cos_thetaj = cosf(rj.theta);
				sin_thetaj = sinf(rj.theta);
				xj = rj.x;
				yj = rj.y;
				zj = rj.z;
				dr = sqrtf(pow(-zi + zj -
			 		R_MON * cos_fii * cos_thetai - R_MON * cos_fij * cos_thetaj,2) +
		   			pow(-xi + xj -
			 		R_MON * (sin_fii * sin_psii + cos_fii * cos_psii * sin_thetai) -
					R_MON * (sin_fij * sin_psij + cos_fij * cos_psij * sin_thetaj),2) +
					pow(-yi + yj -
					R_MON * (-cos_psii * sin_fii + cos_fii * sin_psii * sin_thetai) -
					R_MON * (-cos_psij * sin_fij + cos_fij * sin_psij * sin_thetaj),2));
				
				gradx = -((-xi + xj - R_MON* (sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
						R_MON* (sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj)));

				grady = -((-yi + yj -
						R_MON* (-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
						R_MON* (-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj)));


				gradz = -((-zi + zj - R_MON* cos_fii*cos_thetai - R_MON* cos_fij*cos_thetaj));


				gradtheta = ( R_MON* (-zi + zj -
							R_MON* cos_fii*cos_thetai - R_MON* cos_fij*cos_thetaj)*cos_fii*sin_thetai -
							R_MON* (-xi + xj -
							R_MON* (sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
							R_MON* (sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj))*cos_fii*cos_psii*cos_thetai -
							R_MON* (-yi + yj - R_MON* (-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
							R_MON* (-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj))*cos_fii*cos_thetai*sin_psii);


				gradpsi = (- R_MON* (-xi + xj -
							R_MON* (sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
							R_MON* (sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj))*(cos_psii*sin_fii - cos_fii*sin_psii*sin_thetai) -
							R_MON* (-yi + yj -
							R_MON* (-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
							R_MON* (-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj))*(sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai));


				gradfi = ( R_MON* (-zi + zj -
						R_MON* cos_fii*cos_thetai - R_MON* cos_fij*cos_thetaj)*cos_thetai*sin_fii -
						R_MON* (-xi + xj -
						R_MON* (sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
						R_MON* (sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj))*(cos_fii*sin_psii - cos_psii*sin_fii*sin_thetai) -
						R_MON* (-yi + yj - R_MON* (-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
						R_MON* (-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj))*(-cos_fii*cos_psii - sin_fii*sin_psii*sin_thetai));



				if (dr == 0) dUdr = 0.0;
				else dUdr = dmorse(c_par.D_long, c_par.A_long, dr) / dr;



#if defined(BARR)
				if (dr != 0) 
	            dUdr += dbarr(c_par.a_barr_long, c_par.r_barr_long, c_par.w_barr_long, dr) / dr;
#endif
	        
				fi.x     += -dUdr*gradx;
				fi.y     += -dUdr*grady;
				fi.z     += -dUdr*gradz;
				fi.fi    += -dUdr*gradfi;
				fi.psi   += -dUdr*gradpsi;
				fi.theta += -dUdr*gradtheta;
				
	            if(dr < ANGLE_CUTOFF )
	            {
	            	
	            	psiji = rj.psi - ri.psi;
	                psiij = - psiji;
	            	thetaji = rj.theta - ri.theta;
	            	thetaij = - thetaji;
	              	fiji = rj.fi - ri.fi;
	                fiij = - fiji;
           /// the angle between dimers is ruled by the dimer closest to the plus end, i.e. if it's GDP, then 0.2 rad, if GTP - 0 rad.
	                int last = (ri.z > rj.z) ? (ind + traj * c_par.Ntot) : (j + traj * c_par.Ntot);
	                float theta0 = (c_top.gtp[last] == 1) ? c_par.theta0_gtp : c_par.theta0_gdp;

	                if(R_MON > 0){

	                    fi.psi   += c_par.B_psi		*	sinf(psiji 		- c_par.psi_0);
	                    fi.fi	 += c_par.B_fi		*	sinf(fiji 		- c_par.fi_0	);
	                    fi.theta += c_par.B_theta	*	sinf(thetaji 	- theta0	);
	                }
	                else{
	                    fi.psi   -= c_par.B_psi		*	sinf(psiij	- 	c_par.psi_0	);
	                    fi.fi	 -= c_par.B_fi		*	sinf(fiij 		- c_par.fi_0	);
	                    fi.theta -= c_par.B_theta	*	sinf(thetaij 	- theta0	);
	                }    
	              	
	            }
	            
			} 

#endif

#if defined(MORSE)
			for(int k = 0; k < c_top.lateralCount[ind + traj * c_par.Ntot]; k++){
				j = c_top.lateral[c_top.maxLateralPerMonomer * c_par.Ntot * traj + ind * c_top.maxLateralPerMonomer + k];//c_top.maxLateralPerMonomer*ind+k];
				
				if(j <= 0){
					j = abs(j);
					if (j == ZERO) {j = 0;}
					xp1 = xp2_def;
					yp1 = yp2_def;
					zp1 = zp2_def;
					xp2 = xp1_def;
					yp2 = yp1_def;
					zp2 = zp1_def;
				} else {
					if (j == ZERO) {j = 0;}
					xp1 = xp1_def;
					yp1 = yp1_def;
					zp1 = zp1_def;
					xp2 = xp2_def;
					yp2 = yp2_def;
					zp2 = zp2_def;
				}

				rj = d_r[j + traj*c_par.Ntot];
				cos_fij = cosf(rj.fi);
				sin_fij = sinf(rj.fi);
				cos_psij = cosf(rj.psi);
				sin_psij = sinf(rj.psi);
				cos_thetaj = cosf(rj.theta);
				sin_thetaj = sinf(rj.theta);
				xj = rj.x;
				yj = rj.y;
				zj = rj.z;
				
				dr = sqrtf(pow(zi - zj + zp2 * cos_fii * cos_thetai -
			 		zp1 * cos_fij * cos_thetaj + yp2 * cos_thetai * sin_fii -
			 		yp1 * cos_thetaj * sin_fij - xp2 * sin_thetai + xp1 * sin_thetaj,2) +
		   			pow(xi - xj - yp2 * cos_fii * sin_psii + zp2 * sin_fii * sin_psii +
			  		yp1 * cos_fij * sin_psij - zp1 * sin_fij * sin_psij +
				 	cos_psii * (xp2 * cos_thetai + zp2 * cos_fii * sin_thetai +
					yp2 * sin_fii * sin_thetai) - cos_psij * (xp1 * cos_thetaj + zp1 * cos_fij * sin_thetaj +
			   		yp1 * sin_fij * sin_thetaj),2) +
					pow(yi - yj - zp2 * cos_psii * sin_fii + zp1 * cos_psij * sin_fij +
					xp2 * cos_thetai * sin_psii - xp1 * cos_thetaj * sin_psij +
					yp2 * sin_fii * sin_psii * sin_thetai +
					cos_fii * (yp2 * cos_psii + zp2 * sin_psii * sin_thetai) -
					yp1 * sin_fij * sin_psij * sin_thetaj - cos_fij * (yp1 * cos_psij +
					zp1 * sin_psij * sin_thetaj),2));

				gradx = -((-xi + xj - xp2*cos_psii*cos_thetai +
					  xp1*cos_psij*cos_thetaj -
					  zp2*(sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
					  yp2*(-cos_fii*sin_psii + cos_psii*sin_fii*sin_thetai) +
					  zp1*(sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj) +
					  yp1*(-cos_fij*sin_psij +
						 cos_psij*sin_fij*sin_thetaj)));


				grady = -((-yi + yj -
					  xp2*cos_thetai*sin_psii + xp1*cos_thetaj*sin_psij -
					  zp2*(-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
					  yp2*(cos_fii*cos_psii + sin_fii*sin_psii*sin_thetai) +
					  zp1*(-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj) +
					  yp1*(cos_fij*cos_psij +
						 sin_fij*sin_psij*sin_thetaj)));


				gradz = -((-zi + zj -
					  zp2*cos_fii*cos_thetai + zp1*cos_fij*cos_thetaj -
					  yp2*cos_thetai*sin_fii + yp1*cos_thetaj*sin_fij +
					  xp2*sin_thetai - xp1*sin_thetaj));


				gradtheta = ((-zi + zj - zp2*cos_fii*cos_thetai + zp1*cos_fij*cos_thetaj - yp2*cos_thetai*sin_fii + yp1*cos_thetaj*sin_fij +
					  xp2*sin_thetai - xp1*sin_thetaj)*(xp2*cos_thetai +
					  zp2*cos_fii*sin_thetai + yp2*sin_fii*sin_thetai) +
					 (-xi + xj - xp2*cos_psii*cos_thetai +
					  xp1*cos_psij*cos_thetaj -
					  zp2*(sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
					  yp2*(-cos_fii*sin_psii + cos_psii*sin_fii*sin_thetai) +
					  zp1*(sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj) +
					  yp1*(-cos_fij*sin_psij + cos_psij*sin_fij*sin_thetaj))*(-zp2*cos_fii*cos_psii*cos_thetai - yp2*cos_psii*cos_thetai*sin_fii +
					  xp2*cos_psii*sin_thetai) +
					(-yi + yj - xp2*cos_thetai*sin_psii +
					  xp1*cos_thetaj*sin_psij -
					  zp2*(-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
					  yp2*(cos_fii*cos_psii + sin_fii*sin_psii*sin_thetai) +
					  zp1*(-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj) +
					  yp1*(cos_fij*cos_psij +
						 sin_fij*sin_psij*sin_thetaj))*(-zp2*cos_fii*cos_thetai*sin_psii - yp2*cos_thetai*sin_fii*sin_psii +
					  xp2*sin_psii*sin_thetai));


				gradpsi = ( (-xi + xj -
					  xp2*cos_psii*cos_thetai + xp1*cos_psij*cos_thetaj -
					  zp2*(sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
					  yp2*(-cos_fii*sin_psii + cos_psii*sin_fii*sin_thetai) +
					  zp1*(sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj) +
					  yp1*(-cos_fij*sin_psij +
						 cos_psij*sin_fij*sin_thetaj))*(xp2*cos_thetai*sin_psii -
					   zp2*(cos_psii*sin_fii - cos_fii*sin_psii*sin_thetai) -
					  yp2*(-cos_fii*cos_psii -
						 sin_fii*sin_psii*sin_thetai)) +
					(-yi + yj - xp2*cos_thetai*sin_psii +
					  xp1*cos_thetaj*sin_psij -
					  zp2*(-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
					  yp2*(cos_fii*cos_psii + sin_fii*sin_psii*sin_thetai) +
					  zp1*(-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj) +
					  yp1*(cos_fij*cos_psij +
						 sin_fij*sin_psij*sin_thetaj))*(-xp2*cos_psii*cos_thetai -
					  zp2*(sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
					  yp2*(-cos_fii*sin_psii +
						 cos_psii*sin_fii*sin_thetai)));


				gradfi = ( (-zi + zj -
					  zp2*cos_fii*cos_thetai + zp1*cos_fij*cos_thetaj -
					  yp2*cos_thetai*sin_fii + yp1*cos_thetaj*sin_fij +
					  xp2*sin_thetai - xp1*sin_thetaj)*(-yp2*cos_fii*cos_thetai +
					  zp2*cos_thetai*sin_fii) +
					(-xi + xj - xp2*cos_psii*cos_thetai +
					  xp1*cos_psij*cos_thetaj -
					  zp2*(sin_fii*sin_psii + cos_fii*cos_psii*sin_thetai) -
					  yp2*(-cos_fii*sin_psii + cos_psii*sin_fii*sin_thetai) +
					  zp1*(sin_fij*sin_psij + cos_fij*cos_psij*sin_thetaj) +
					  yp1*(-cos_fij*sin_psij +
						 cos_psij*sin_fij*sin_thetaj))*(-yp2*(sin_fii*sin_psii +
						 cos_fii*cos_psii*sin_thetai) -
					  zp2*(cos_fii*sin_psii -
						 cos_psii*sin_fii*sin_thetai)) +
					(-yi + yj - xp2*cos_thetai*sin_psii +
					  xp1*cos_thetaj*sin_psij -
					  zp2*(-cos_psii*sin_fii + cos_fii*sin_psii*sin_thetai) -
					  yp2*(cos_fii*cos_psii + sin_fii*sin_psii*sin_thetai) +
					  zp1*(-cos_psij*sin_fij + cos_fij*sin_psij*sin_thetaj) +
					  yp1*(cos_fij*cos_psij +
						 sin_fij*sin_psij*sin_thetaj))*(-yp2*(-cos_psii*sin_fii +
						  cos_fii*sin_psii*sin_thetai) -
					  zp2*(-cos_fii*cos_psii -
						 sin_fii*sin_psii*sin_thetai)));



				if (dr == 0) dUdr = 0.0;
				else if (c_top.mon_type[ind] != c_top.mon_type[j]) {
					dUdr = dmorse(c_par.D_lat / 2, c_par.A_lat, dr) / dr;
				}
	            else {
	            	dUdr = dmorse(c_par.D_lat, c_par.A_lat, dr) / dr;
	            }
	            

#if defined(BARR)
			if (dr != 0) 
            dUdr += dbarr(c_par.a_barr_long, c_par.r_barr_long, c_par.w_barr_long, dr) / dr;
#endif

				fi.x     += -dUdr*gradx;
				fi.y     += -dUdr*grady;
				fi.z     += -dUdr*gradz;
				fi.fi    += -dUdr*gradfi;
				fi.psi   += -dUdr*gradpsi;
				fi.theta += -dUdr*gradtheta;
			
			}
#endif


			if (c_par.lj_on) {

				for(int k = 0; k < c_top.LJCount[ind + traj * c_par.Ntot]; k++){

					j = c_top.LJ[c_top.maxLJPerMonomer * c_par.Ntot * traj + ind * c_top.maxLJPerMonomer + k];
					rj = d_r[j + traj * c_par.Ntot];
					dr = sqrt(pow(ri.x-rj.x,2)+pow(ri.y-rj.y,2)+pow(ri.z-rj.z,2));
					
					if( dr < lj_cutoff )
		            {	
		            	real df = 6/pow(dr,8);

		            #ifdef REGULARIZATION
		            	if (c_par.ljscale * c_par.ljsigma6 * df * dr > c_par.gammaR * r_mon / c_par.dt ){
		            		df = c_par.gammaR * r_mon / (c_par.dt * dr * c_par.ljscale * c_par.ljsigma6);	
		            	} 
					#endif	
		            	
		            	fi.x += c_par.ljscale*c_par.ljsigma6*df*(ri.x-rj.x);
						fi.y += c_par.ljscale*c_par.ljsigma6*df*(ri.y-rj.y);
						fi.z += c_par.ljscale*c_par.ljsigma6*df*(ri.z-rj.z); 

					}
				}

			}

			if (c_par.is_wall) {

				if (ri.z < c_par.rep_leftborder){ 

		        	fi.z += c_par.rep_eps * fabs(ri.z - c_par.rep_leftborder);
		    	} else if (ri.z > c_par.zs[traj] + c_par.rep_leftborder) {

		    	fi.z += - c_par.rep_eps * fabs(ri.z - (c_par.zs[traj] + c_par.rep_leftborder) );
		   		}

			    real rad2 = ri.x * ri.x + ri.y * ri.y;

			    if (rad2 > c_par.rep_r * c_par.rep_r){

			        real coeff = -c_par.rep_eps * (sqrt(rad2) - c_par.rep_r);
			        fi.x += ri.x * coeff ;
			        fi.y += ri.y * coeff;
			    }
			} 	
		    

			d_f[p] = fi;
			fi = (Coord){0.0,0.0,0.0,0.0,0.0,0.0};
			
		}
	}
}

__global__ void pairs_kernel(const Coord* d_r){
	const int p = blockIdx.x * blockDim.x + threadIdx.x;
	const int i = p % c_par.Ntot;
	const int traj = p / c_par.Ntot;
	real cos_fii, cos_fij, sin_fii, sin_fij, 
		  cos_psii, cos_psij, sin_psii, sin_psij,
		  cos_thetai, cos_thetaj, sin_thetai, sin_thetaj;
	real xi, xj, yi, yj, zi, zj;
	real dr, dr2;
    Coord ri, rj;
    real xp1, yp1, zp1, xp2, yp2, zp2;
    real R_MON;
    
    if(i < c_par.Ntot && traj < c_par.Ntr){

    	c_top.lateralCount[i + traj * c_par.Ntot] = 0;
    	c_top.longitudinalCount[i + traj * c_par.Ntot] = 0;

    	if(!c_top.extra[i + traj * c_par.Ntot]){
			
			
			if(c_top.harmonic[c_top.maxHarmonicPerMonomer * i ] < 0) /// c_top.maxHarmonicPerMonomer*ind+k
		        R_MON = -r_mon; 
		    else
		        R_MON = r_mon;
		        
		    ri = d_r[p];
		    cos_fii = cosf(ri.fi); 
		    sin_fii = sinf(ri.fi);
		    cos_psii = cosf(ri.psi);
		    sin_psii = sinf(ri.psi);
		    cos_thetai = cosf(ri.theta);
		    sin_thetai = sinf(ri.theta);
		    xi = ri.x;
		    yi = ri.y;
		    zi = ri.z;

		    for(int j = 0; j < c_par.Ntot; j++){
		        if(c_top.mon_type[i] != c_top.mon_type[j] && abs(c_top.harmonic[c_top.maxHarmonicPerMonomer * i]) != j ){ // C_top.mon_type
		            rj = d_r[j + traj * c_par.Ntot];
		            cos_fij = cosf(rj.fi);
		            sin_fij = sinf(rj.fi);
		            cos_psij = cosf(rj.psi);
		            sin_psij = sinf(rj.psi);
		            cos_thetaj = cosf(rj.theta);
		            sin_thetaj = sinf(rj.theta);
		            xj = rj.x;
		            yj = rj.y;
		            zj = rj.z;
		            dr2 = pow(-zi + zj -
		                R_MON * cos_fii * cos_thetai - R_MON * cos_fij * cos_thetaj,2) +
		                pow(-xi + xj -
		                R_MON * (sin_fii * sin_psii + cos_fii * cos_psii * sin_thetai) -
		                R_MON * (sin_fij * sin_psij + cos_fij * cos_psij * sin_thetaj),2) +
		                pow(-yi + yj -
		                R_MON * (-cos_psii * sin_fii + cos_fii * sin_psii * sin_thetai) -
		                R_MON * (-cos_psij * sin_fij + cos_fij * sin_psij * sin_thetaj),2);
		            dr  = sqrt(dr2);

		            if(dr < PAIR_CUTOFF){
		            	
		            	if(c_top.harmonic[c_top.maxHarmonicPerMonomer * i] < 0)
		                    c_top.longitudinal[c_top.maxLongitudinalPerMonomer * c_par.Ntot * traj + i * c_top.maxLongitudinalPerMonomer + c_top.longitudinalCount[i + traj * c_par.Ntot]] = j;
		                else
		                    c_top.longitudinal[c_top.maxLongitudinalPerMonomer * c_par.Ntot * traj + i * c_top.maxLongitudinalPerMonomer + c_top.longitudinalCount[i + traj * c_par.Ntot]] = -j;
		                c_top.longitudinalCount[i + traj * c_par.Ntot]++;
		            }
		            
		        }
		    }
		    
		    for(int j = 0; j < c_par.Ntot; j++){
		        if (i != j && abs(c_top.harmonic[c_top.maxHarmonicPerMonomer * i]) != j){// && c_top.mon_type[i] == c_top.mon_type[j]) {

		        	rj = d_r[j + traj * c_par.Ntot];
		            xj = rj.x;    
		            yj = rj.y;    
		            zj = rj.z;
		            sin_fij = sinf(rj.fi);
		            cos_fij = cosf(rj.fi);
		            sin_psij = sinf(rj.psi);
		            cos_psij = cosf(rj.psi);      
		            sin_thetaj = sinf(rj.theta);
		            cos_thetaj = cosf(rj.theta); 
		             
		            for(int ind = 0; ind < 2; ind++){   
		                if(ind == 0){
		                    xp1 = xp2_def;
		                    yp1 = yp2_def;
		                    zp1 = zp2_def;
		                    xp2 = xp1_def;
		                    yp2 = yp1_def;
		                    zp2 = zp1_def;
		                } else {
		                    xp1 = xp1_def;
		                    yp1 = yp1_def;
		                    zp1 = zp1_def;
		                    xp2 = xp2_def;
		                    yp2 = yp2_def;
		                    zp2 = zp2_def;
		                }

		                dr = sqrtf(pow(zi - zj + zp2 * cos_fii * cos_thetai -
		                zp1 * cos_fij * cos_thetaj + yp2 * cos_thetai * sin_fii -
		                yp1 * cos_thetaj * sin_fij - xp2 * sin_thetai + xp1 * sin_thetaj,2) +
		                pow(xi - xj - yp2 * cos_fii * sin_psii + zp2 * sin_fii * sin_psii +
		                yp1 * cos_fij * sin_psij - zp1 * sin_fij * sin_psij +
		                cos_psii * (xp2 * cos_thetai + zp2 * cos_fii * sin_thetai +
		                yp2 * sin_fii * sin_thetai) - cos_psij * (xp1 * cos_thetaj + zp1 * cos_fij * sin_thetaj +
		                yp1 * sin_fij * sin_thetaj),2) + pow(yi - yj - zp2 * cos_psii * sin_fii + zp1 * cos_psij * sin_fij +
		                xp2 * cos_thetai * sin_psii - xp1 * cos_thetaj * sin_psij + yp2 * sin_fii * sin_psii * sin_thetai +
		                cos_fii * (yp2 * cos_psii + zp2 * sin_psii * sin_thetai) -
		                yp1 * sin_fij * sin_psij * sin_thetaj - cos_fij * (yp1 * cos_psij +
		                zp1 * sin_psij * sin_thetaj),2));
						
						

						if (dr < PAIR_CUTOFF){
							if (ind == 0){ 
								if (j != 0) {
									c_top.lateral[c_top.maxLateralPerMonomer * c_par.Ntot * traj + i * c_top.maxLateralPerMonomer + c_top.lateralCount[i + traj * c_par.Ntot]] = -j;
								} else {
									c_top.lateral[c_top.maxLateralPerMonomer * c_par.Ntot * traj + i * c_top.maxLateralPerMonomer + c_top.lateralCount[i + traj * c_par.Ntot]] = -ZERO;
								}
							c_top.lateralCount[i + traj * c_par.Ntot]++;

							} else {//if (c_top.lateral[c_top.maxLateralPerMonomer * c_par.Ntot * traj + i * c_top.maxLateralPerMonomer + c_top.lateralCount[i + traj * c_par.Ntot] - 1] + j != 0){
								if (j != 0){
									c_top.lateral[c_top.maxLateralPerMonomer * c_par.Ntot * traj + i * c_top.maxLateralPerMonomer + c_top.lateralCount[i + traj * c_par.Ntot]] = j;
								} else {
									c_top.lateral[c_top.maxLateralPerMonomer * c_par.Ntot * traj + i * c_top.maxLateralPerMonomer + c_top.lateralCount[i + traj * c_par.Ntot]] = ZERO;
								}
								
							c_top.lateralCount[i + traj * c_par.Ntot]++;
							}
		 	
						}

					}
					
		       } 

		    }
		    
		}   
    } 
    
}

__global__ void energy_kernel(const Coord* d_r, Energies* d_energies){
	
	const int p = blockIdx.x * blockDim.x + threadIdx.x;
	const int ind = p % c_par.Ntot;
	const int traj = p / c_par.Ntot;
	real cos_fii, cos_fij, sin_fii, sin_fij, 
		  cos_psii, cos_psij, sin_psii, sin_psij,
		  cos_thetai, cos_thetaj, sin_thetai, sin_thetaj;
	real xi, xj, yi, yj, zi, zj;
	real psiji, thetaji, fiji, psiij, thetaij, fiij;
	int j;
	Coord ri, rj;
	real dr, dr2;
	Energies en;
	real U_lat = 0.0, U_long = 0.0, U_harm = 0.0, U_fi = 0.0, U_psi = 0.0, U_teta = 0.0, U_lj = 0.0;

	real xp1 = xp1_def;
	real yp1 = yp1_def;
	real zp1 = zp1_def;
	real xp2 = xp2_def;
	real yp2 = yp2_def;
	real zp2 = zp2_def;
	real R_MON = r_mon;

	if (ind < c_par.Ntot && traj < c_par.Ntr){
		if(!c_top.extra[ind + traj * c_par.Ntot]){
			ri = d_r[p];
			cos_fii = cosf(ri.fi); 
			sin_fii = sinf(ri.fi);
			cos_psii = cosf(ri.psi);
			sin_psii = sinf(ri.psi);
			cos_thetai = cosf(ri.theta);
			sin_thetai = sinf(ri.theta);
			xi = ri.x;
			yi = ri.y;
			zi = ri.z;
			
			for(int k = 0; k < c_top.harmonicCount[ind]; k++){
				j = c_top.harmonic[c_top.maxHarmonicPerMonomer * ind + k];
				if (j < 0){
					R_MON = r_mon;
					j *= -1;					
				} else {
					R_MON = -r_mon;
				}
				rj = d_r[j + traj * c_par.Ntot];
				cos_fij = cosf(rj.fi);
				sin_fij = sinf(rj.fi);
				cos_psij = cosf(rj.psi);
				sin_psij = sinf(rj.psi);
				cos_thetaj = cosf(rj.theta);
				sin_thetaj = sinf(rj.theta);
				xj = rj.x;
				yj = rj.y;
				zj = rj.z;
				dr = sqrt(pow(-zi + zj -
					R_MON * cos_fii * cos_thetai - R_MON * cos_fij * cos_thetaj,2) +
					pow(-xi + xj -
					R_MON * (sin_fii * sin_psii + cos_fii * cos_psii * sin_thetai) -
					R_MON * (sin_fij * sin_psij + cos_fij * cos_psij * sin_thetaj),2) +
					pow(-yi + yj -
					R_MON * (-cos_psii * sin_fii + cos_fii * sin_psii * sin_thetai) -
					R_MON * (-cos_psij * sin_fij + cos_fij * sin_psij * sin_thetaj),2));

				U_harm += (c_par.C / 2) * pow(dr,2);

	            if(dr < ANGLE_CUTOFF){

	            	psiji = rj.psi - ri.psi;
	                psiij = - psiji;
	            	thetaji = rj.theta - ri.theta;
	            	thetaij = - thetaji;
	              	fiji = rj.fi - ri.fi;
	                fiij = - fiji;

	              	U_psi  	 += c_par.B_psi	  	* (1 - cosf(psiij 		- c_par.psi_0))		;
	                U_fi	 += c_par.B_fi	  	* (1 - cosf(fiij 		- c_par.fi_0))		;
	                if (c_top.gtp[p] == 1){
	                    U_teta	 += c_par.B_theta	* (1 - cosf(thetaji 	- c_par.theta0_gtp))   ;
	                } else {
	                    U_teta	 += c_par.B_theta	* (1 - cosf(thetaji 	- c_par.theta0_gdp))   ;
	                }
	                
	                    
	            }
			}

#if defined(MORSE)		
			for(int k = 0; k < c_top.longitudinalCount[ind + traj * c_par.Ntot]; k++){
				j = c_top.longitudinal[c_top.maxLongitudinalPerMonomer * c_par.Ntot * traj + c_top.maxLongitudinalPerMonomer * ind + k];
				if (j < 0) {
					R_MON = r_mon;
					j *= -1;
				}
				else{
					R_MON = -r_mon;
				}
				rj = d_r[j + traj*c_par.Ntot];
				cos_fij = cosf(rj.fi);
				sin_fij = sinf(rj.fi);
				cos_psij = cosf(rj.psi);
				sin_psij = sinf(rj.psi);
				cos_thetaj = cosf(rj.theta);
				sin_thetaj = sinf(rj.theta);
				xj = rj.x;
				yj = rj.y;
				zj = rj.z;
				dr2 = pow(-zi + zj -
					R_MON * cos_fii * cos_thetai - R_MON * cos_fij * cos_thetaj,2) +
					pow(-xi + xj -
					R_MON * (sin_fii * sin_psii + cos_fii * cos_psii * sin_thetai) -
					R_MON * (sin_fij * sin_psij + cos_fij * cos_psij * sin_thetaj),2) +
					pow(-yi + yj -
					R_MON * (-cos_psii * sin_fii + cos_fii * sin_psii * sin_thetai) -
					R_MON * (-cos_psij * sin_fij + cos_fij * sin_psij * sin_thetaj),2);
				dr	= sqrt(dr2);


	            U_long += morse_en(c_par.D_long, c_par.A_long, dr);
				//U_long += (c_par.A_long*(c_par.b_long * dr2 * exp(-dr / c_par.r0_long) - c_par.c_long*exp(-dr2/(c_par.d_long*c_par.r0_long)))); 


#if defined(BARR)
	            U_long += barr(c_par.a_barr_long, c_par.r_barr_long, c_par.w_barr_long, dr);
#endif
				if(dr < ANGLE_CUTOFF){

	            	psiji = rj.psi - ri.psi;
	                psiij = - psiji;
	            	thetaji = rj.theta - ri.theta;
	            	thetaij = - thetaji;
	              	fiji = rj.fi - ri.fi;
	                fiij = - fiji;

	                int last = (ri.z > rj.z) ? (ind + traj * c_par.Ntot) : (j + traj * c_par.Ntot);
	                float theta0 = (c_top.gtp[last] == 1) ? c_par.theta0_gtp : c_par.theta0_gdp;

	             	U_psi  	 += c_par.B_psi	  	* (1 - cosf(psiij 		- c_par.psi_0))		;
					U_fi	 += c_par.B_fi	  	* (1 - cosf(fiij 		- c_par.fi_0))		;
	                U_teta	 += c_par.B_theta	* (1 - cosf(thetaij 	- theta0))   ;
	            }
			}
#endif

#if defined(MORSE)
			for(int k = 0; k < c_top.lateralCount[ind + traj * c_par.Ntot]; k++){
				j = c_top.lateral[c_top.maxLateralPerMonomer * c_par.Ntot * traj + c_top.maxLateralPerMonomer * ind + k];
				
				if (j <= 0){
					j = abs(j);
					if (j == ZERO) {j = 0;}
					xp1 = xp2_def;
					yp1 = yp2_def;
					zp1 = zp2_def;
					xp2 = xp1_def;
					yp2 = yp1_def;
					zp2 = zp1_def;
				}
				else
				{
					if (j == ZERO) {j = 0;}
					xp1 = xp1_def;
					yp1 = yp1_def;
					zp1 = zp1_def;
					xp2 = xp2_def;
					yp2 = yp2_def;
					zp2 = zp2_def;
				}

				rj = d_r[j + traj * c_par.Ntot];
				cos_fij = cosf(rj.fi);
				sin_fij = sinf(rj.fi);
				cos_psij = cosf(rj.psi);
				sin_psij = sinf(rj.psi);
				cos_thetaj = cosf(rj.theta);
				sin_thetaj = sinf(rj.theta);
				xj = rj.x;
				yj = rj.y;
				zj = rj.z;
				
				dr = sqrt(pow(zi - zj + zp2 * cos_fii * cos_thetai -
					zp1 * cos_fij * cos_thetaj + yp2 * cos_thetai * sin_fii -
					yp1 * cos_thetaj * sin_fij - xp2 * sin_thetai + xp1 * sin_thetaj,2) +
					pow(xi - xj - yp2 * cos_fii * sin_psii + zp2 * sin_fii * sin_psii +
					yp1 * cos_fij * sin_psij - zp1 * sin_fij * sin_psij +
					cos_psii * (xp2 * cos_thetai + zp2 * cos_fii * sin_thetai +
					yp2 * sin_fii * sin_thetai) - cos_psij * (xp1 * cos_thetaj + zp1 * cos_fij * sin_thetaj +
					yp1 * sin_fij * sin_thetaj),2) +
					pow(yi - yj - zp2 * cos_psii * sin_fii + zp1 * cos_psij * sin_fij +
					xp2 * cos_thetai * sin_psii - xp1 * cos_thetaj * sin_psij +
					yp2 * sin_fii * sin_psii * sin_thetai +
					cos_fii * (yp2 * cos_psii + zp2 * sin_psii * sin_thetai) -
					yp1 * sin_fij * sin_psij * sin_thetaj - cos_fij * (yp1 * cos_psij +
					zp1 * sin_psij * sin_thetaj),2));
				dr2 = dr*dr;
			

	            if (c_top.mon_type[ind] != c_top.mon_type[j]) {
					U_lat += morse_en(c_par.D_lat / 2, c_par.A_lat, dr); 	
				}
	            else {
	            	U_lat += morse_en(c_par.D_lat, c_par.A_lat, dr);
	            }
				


#if defined(BARR)
	            U_lat += barr(c_par.a_barr_lat, c_par.r_barr_lat, c_par.w_barr_lat, dr);
#endif
			}
#endif
		

			if (c_par.lj_on){
				for(int k = 0; k < c_top.LJCount[ind + traj * c_par.Ntot]; k++){
		        	j = c_top.LJ[c_top.maxLJPerMonomer * c_par.Ntot * traj + ind * c_top.maxLJPerMonomer + k];
					rj = d_r[j + traj * c_par.Ntot];
		            dr = sqrt(pow(ri.x - rj.x, 2) + pow(ri.y - rj.y, 2) + pow(ri.z - rj.z, 2));
		            if(dr < lj_cutoff){
		                U_lj += c_par.ljscale * c_par.ljsigma6 / pow(dr,6);
		            }
		        }
			}       

		}

		en.U_harm = U_harm / 2;
		en.U_long = U_long / 2;
		en.U_lat = U_lat / 2;
		en.U_lj = U_lj / 2;
		en.U_psi = U_psi / 2;
		en.U_fi = U_fi / 2;
		en.U_teta = U_teta / 2;

		d_energies[p] = en; 	
	}
}

__global__ void LJ_kernel(const Coord* r){

    const int p = blockIdx.x*blockDim.x + threadIdx.x;
    const int i = p % c_par.Ntot;
    const int tr = p/c_par.Ntot;
    Coord ri, rj;
    
    if(i < c_par.Ntot && tr < c_par.Ntr){

    	ri = r[i + tr * c_par.Ntot];
	    c_top.LJCount[i + tr * c_par.Ntot] = 0;

    	if(!c_top.extra[i + tr * c_par.Ntot]){

	        for(int j = 0; j < c_par.Ntot; j++){
	            rj = r[j + tr * c_par.Ntot];
	            real dr = sqrt(pow(ri.x - rj.x,2)+
	                            pow(ri.y - rj.y,2)+
	                            pow(ri.z - rj.z,2));

	            if((dr < c_par.ljpairscutoff) && (i != j)){
	                c_top.LJCount[i + tr * c_par.Ntot]++;
	                c_top.LJ[c_top.maxLJPerMonomer * c_par.Ntot * tr + i * c_top.maxLJPerMonomer + c_top.LJCount[i + tr * c_par.Ntot] - 1] = j;
	            }
        	}   
    	}
	}
}


__global__ void integrate_kernel(Coord* d_r, Coord* d_f){
	const int p = blockIdx.x*blockDim.x + threadIdx.x;
	float4 rf_xyz = make_float4(0,0,0,0);
	float4 rf_ang = make_float4(0,0,0,0);
	if(p < c_par.Ntot * c_par.Ntr){
		Coord f, ri;
		if(!c_top.fixed[p % c_par.Ntot] && !(c_top.extra[p])){
			f = d_f[p];
			ri = d_r[p];
			rf_xyz = rforce(p);
			rf_ang = rforce(p + c_par.Ntot*c_par.Ntr);

			ri.x += (c_par.dt/c_par.gammaR)*f.x + c_par.varR*rf_xyz.x;
			ri.y += (c_par.dt/c_par.gammaR)*f.y + c_par.varR*rf_xyz.y;
			ri.z += (c_par.dt/c_par.gammaR)*f.z + c_par.varR*rf_xyz.z;

			ri.fi    += (c_par.dt/(c_par.gammaTheta * c_par.alpha))*f.fi  + (c_par.varTheta * sqrt(c_par.freeze_temp / c_par.alpha))*rf_ang.x;
			ri.psi   += (c_par.dt/(c_par.gammaTheta * c_par.alpha))*f.psi + (c_par.varTheta * sqrt(c_par.freeze_temp / c_par.alpha))*rf_ang.y;
			ri.theta += (c_par.dt/c_par.gammaTheta)*f.theta + c_par.varTheta*rf_ang.z;

			d_r[p] = ri;
		}
		
		f.x = 0.0f;
		f.y = 0.0f;
		f.z = 0.0f;
		f.fi = 0.0f;
		f.psi = 0.0f;
		f.theta = 0.0f;
		d_f[p] = f;
		
	}
}

void initIntegration(Coord* r, Coord* f, Parameters &par, Topology &top, Energies* energies){
	//printf("device is %d\n", par.device);
	hipSetDevice(par.device);
	hipGetDevice(&(par.device));
	hipDeviceProp_t deviceProp; 

	hipGetDeviceProperties(&deviceProp, par.device);

	printf("Using device %d: %s \n", par.device, deviceProp.name);

	checkCUDAError("device: mistake");

	hipMalloc((void**)&d_r, par.Ntot*par.Ntr*sizeof(Coord));
	checkCUDAError("d_r allocation");
	hipMalloc((void**)&d_f, par.Ntot*par.Ntr*sizeof(Coord));
	checkCUDAError("d_f allocation");

	// security from jerk
	for(int i = 0; i < par.Ntot*par.Ntr; i++){
		f[i].fi = 0.0f;
		f[i].psi = 0.0f;
		f[i].theta = 0.0f;
		f[i].x = 0.0f;
		f[i].y = 0.0f;
		f[i].z = 0.0f;
	}

	for(int i = 0; i < par.Ntot*par.Ntr; i++){
		
		r[i].fi -= (2 * M_PI) * (int)(r[i].fi / (2 * M_PI));
        r[i].psi -= (2 * M_PI) * (int)(r[i].psi / (2 * M_PI));
        r[i].theta -= (2 * M_PI) * (int)(r[i].theta / (2 * M_PI));
        
	}

	hipMemcpy(d_f, f, par.Ntot*par.Ntr*sizeof(Coord), hipMemcpyHostToDevice);
	checkCUDAError("copy_forces");
	hipMemcpy(d_r, r, par.Ntot*par.Ntr*sizeof(Coord), hipMemcpyHostToDevice);
	checkCUDAError("from r to d_r copy");

	topGPU.maxHarmonicPerMonomer = top.maxHarmonicPerMonomer;
	topGPU.maxLongitudinalPerMonomer = top.maxLongitudinalPerMonomer;
	topGPU.maxLateralPerMonomer = top.maxLateralPerMonomer;

	hipMalloc((void**)&(topGPU.harmonicCount), par.Ntot*sizeof(int));
	checkCUDAError("topGPU.harmonicCount allocation");
	hipMalloc((void**)&(topGPU.longitudinalCount), par.Ntot*par.Ntr*sizeof(int));
	checkCUDAError("topGPU.longitudinalCount allocation");
	hipMalloc((void**)&(topGPU.lateralCount), par.Ntot*par.Ntr*sizeof(int));
	checkCUDAError("topGPU.lateralCount allocation");
	hipMalloc((void**)&(topGPU.harmonic), par.Ntot*sizeof(int)*topGPU.maxHarmonicPerMonomer);
	checkCUDAError("harmonic allocation");
	hipMalloc((void**)&(topGPU.longitudinal), par.Ntot*par.Ntr*sizeof(int)*topGPU.maxLongitudinalPerMonomer);
	checkCUDAError("long allocation");
	hipMalloc((void**)&(topGPU.lateral), par.Ntot*par.Ntr*sizeof(int)*topGPU.maxLateralPerMonomer);
	checkCUDAError("lateral allocation");

	hipMemcpy(topGPU.harmonic, top.harmonic, par.Ntot*topGPU.maxHarmonicPerMonomer*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("harmonic copy");
	hipMemcpy(topGPU.longitudinal, top.longitudinal, par.Ntot*par.Ntr*topGPU.maxLongitudinalPerMonomer*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("long copy");
	hipMemcpy(topGPU.lateral, top.lateral, par.Ntot*par.Ntr*topGPU.maxLateralPerMonomer*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("lateral copy");
	hipMemcpy(topGPU.harmonicCount, top.harmonicCount, par.Ntot*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("harmonic count copy");
	hipMemcpy(topGPU.longitudinalCount, top.longitudinalCount, par.Ntot*par.Ntr*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("long count copy");
	hipMemcpy(topGPU.lateralCount, top.lateralCount, par.Ntot*par.Ntr*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("lat count copy");

	hipMalloc((void**)&(topGPU.fixed), par.Ntot*sizeof(bool));
	checkCUDAError("topGPU.fixed allocation");
	hipMalloc((void**)&(topGPU.mon_type), par.Ntot*sizeof(int));
	checkCUDAError("d_mon_type allocation");
	hipMalloc((void**)&(topGPU.extra), par.Ntr * par.Ntot*sizeof(bool));
	checkCUDAError("topGPU.extra allocation");
	hipMalloc((void**)&(topGPU.gtp), par.Ntr * par.Ntot*sizeof(int));
	checkCUDAError("topGPU.gtp allocation");
	hipMalloc((void**)&(topGPU.on_tubule), par.Ntr * par.Ntot*sizeof(int));
	checkCUDAError("topGPU.extra allocation");

	hipMemcpy(topGPU.fixed, top.fixed, par.Ntot*sizeof(bool), hipMemcpyHostToDevice);
	checkCUDAError("fixed copy");
	hipMemcpy(topGPU.mon_type, top.mon_type, par.Ntot*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("montype copy");
	hipMemcpy(topGPU.extra, top.extra, par.Ntr * par.Ntot*sizeof(bool), hipMemcpyHostToDevice);
	checkCUDAError("extra copy");
	hipMemcpy(topGPU.gtp, top.gtp, par.Ntr * par.Ntot*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("gtp copy");
	hipMemcpy(topGPU.on_tubule, top.on_tubule, par.Ntr * par.Ntot*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("on tubule copy");

	if (par.lj_on){
		topGPU.maxLJPerMonomer = 256;  ///I hope it will be enough       =top.maxLJPerMonomer;
		hipMalloc((void**)&(topGPU.LJCount), par.Ntot*sizeof(int)*par.Ntr);
		checkCUDAError("lj_count allocation");
		hipMalloc((void**)&(topGPU.LJ), par.Ntot*sizeof(int)*par.Ntr*topGPU.maxLJPerMonomer);
		checkCUDAError("lj allocation");	
	}
	

	//const memory
	hipMemcpyToSymbol(HIP_SYMBOL(c_top), &topGPU, sizeof(Topology), 0, hipMemcpyHostToDevice);
	checkCUDAError("copy of topGPU pointer to const memory");

	hipMemcpyToSymbol(HIP_SYMBOL(c_par), &par, sizeof(Parameters), 0, hipMemcpyHostToDevice);
	checkCUDAError("copy parameters to const memory");

    //energies initializing
    if (par.out_energy) {	
		hipMalloc((void**)&d_energies, par.Ntot*par.Ntr * sizeof(Energies));
		checkCUDAError("d_energies allocation");
    }

	initRand(par.rseed, 2*par.Ntot*par.Ntr);
}

void deleteIntegration(Coord* r, Coord* f, Parameters &par, Topology &top, Energies* energies){
	hipFree(d_r);
	hipFree(d_f);

	hipFree(topGPU.harmonicCount);
	hipFree(topGPU.longitudinalCount);
	hipFree(topGPU.lateralCount);
	hipFree(topGPU.fixed);
	hipFree(topGPU.mon_type);
	hipFree(topGPU.extra);
	hipFree(topGPU.harmonic);
	hipFree(topGPU.longitudinal);
	hipFree(topGPU.lateral);
	if (par.lj_on){
		hipFree(topGPU.LJCount);
		hipFree(topGPU.LJ);
	} 
	if (par.out_energy)
		hipFree(d_energies);

	checkCUDAError("cleanup");
}

void compute(Coord* r, Coord* f, Parameters &par, Topology &top, Energies* energies){

	initIntegration(r, f, par, top, energies);

	if (par.hdi_on) {
		initTeaIntegrator();																	
	} 

    int* mt_len = (int*)malloc(par.Ntr * sizeof(int));
	int* mt_len_prev = (int*)malloc(par.Ntr * sizeof(int));
	//int* delta = (int*)malloc(par.Ntr * sizeof(int))

	for(long long int step = 0; step < par.steps; step++){				// <-- Start simulations


		if(step % par.ljpairsupdatefreq == 0){ //pairs update frequency 

			if (par.lj_on){
				LJ_kernel<<<par.Ntot*par.Ntr/BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_r);
				checkCUDAError("lj_kernel");
			}

			if (par.is_assembly){
				pairs_kernel<<<par.Ntot*par.Ntr/BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_r);
            	checkCUDAError("pairs_kernel");
			}
		}

		if (par.hydrolysis){
			if(step % par.hydrostep == 0){
				//printf("Making hydrolysis: step = %ld\n", par.hydrostep);
				hydrolyse();
				hipMemcpy(topGPU.gtp, top.gtp, par.Ntr * par.Ntot*sizeof(int), hipMemcpyHostToDevice);
				checkCUDAError("gtp copy");        
			}
		}
		

		if(step % par.stride == 0){ //every stride steps do energy computing / outputing DCD / mt_length measurements  / const concentration
		
			if (par.out_energy) {
	            energy_kernel<<<par.Ntot*par.Ntr/BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_r, d_energies);
	            checkCUDAError("energy_kernel");
	            hipMemcpy(energies, d_energies, par.Ntr * par.Ntot * sizeof(Energies), hipMemcpyDeviceToHost);
	            checkCUDAError("energy_copy");
	        }

			hipMemcpy(r, d_r, par.Ntr*par.Ntot*sizeof(Coord), hipMemcpyDeviceToHost);
			checkCUDAError("r copy");

			if (par.out_force) {
				hipMemcpy(f, d_f, par.Ntot*par.Ntr*sizeof(Coord), hipMemcpyDeviceToHost);
				checkCUDAError("forces copy");
			}



			if (par.tub_length) {
				if (step != 0){

					memcpy(mt_len_prev, mt_len, par.Ntr*sizeof(int));
					mt_length(step, mt_len);
							
					if (par.is_const_conc){
						for (int i = 0; i < par.Ntr; i++){
							mt_len_prev[i] = mt_len[i] - mt_len_prev[i];
						}

						if (change_conc(mt_len_prev, mt_len)){
							hipMemcpy(topGPU.extra, top.extra, par.Ntr * par.Ntot*sizeof(bool), hipMemcpyHostToDevice);
							checkCUDAError("extra copy to device");
							hipMemcpy(d_r, r, par.Ntot*par.Ntr*sizeof(Coord), hipMemcpyHostToDevice);
							checkCUDAError("from r to d_r copy");
							//hipMemcpyToSymbol(HIP_SYMBOL(c_par), &par, sizeof(Parameters), 0, hipMemcpyHostToDevice);
							//checkCUDAError("copy parameters to const memory");

							//hipMemcpyToSymbol(HIP_SYMBOL(c_top), &topGPU, sizeof(Topology), 0, hipMemcpyHostToDevice);
							//checkCUDAError("copy of topGPU pointer to const memory");
						}
					}
		
					update(step, mt_len);	
					
					} 
					else{
						update(step, mt_len);
						mt_length(step, mt_len);
					}
				} else {
					update(step, mt_len);
				}			
			
		}

		compute_kernel<<<par.Ntot*par.Ntr/BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_r, d_f);
		checkCUDAError("compute_kernel");
		
		if (par.hdi_on) {
			updateTea(step);
			integrateTea();

		} else {
			integrate_kernel<<<par.Ntot*par.Ntr/BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_r, d_f);
			checkCUDAError("integrate_kernel");
		}
		

		/*
		hipMemcpy(top.longitudinal, topGPU.longitudinal, par.Ntot*par.Ntr*top.maxLongitudinalPerMonomer*sizeof(int), hipMemcpyDeviceToHost);
		printf("Done\n");
		hipMemcpy(top.longitudinalCount, topGPU.longitudinalCount, par.Ntot*par.Ntr*sizeof(int), hipMemcpyDeviceToHost);
		

		for (int i = 0; i < par.Ntot; i++){
			//c_top.lateral[c_top.maxLateralPerMonomer * c_par.Ntot * traj + c_top.maxLateralPerMonomer * ind + k];
			if (top.longitudinalCount[i] > 0)
			printf("[%d] %d\n", i, top.longitudinal[ top.maxLongitudinalPerMonomer * i + 0]);
		}
		*/
	}

	if (par.hdi_on) {
		deleteTeaIntegrator();																	
		//Hydrodynamics integrator;
	} 
	deleteIntegration(r, f, par, top, energies);
}

