#include "hip/hip_runtime.h"
/*
 * bdhitea.cu
 *
 *  Created on: Mar 01, 2013
 *	  Author: alekseenko
 * 
 * The description of algorithm is available in Geyer & Winter, 2009 [doi:10.1063/1.3089668]
 * All the equations referenced here are from the very same paper
 */

#include "bdhitea.cuh"

void initTeaIntegrator(){
	
	//initLangevinIntegrator();
	//teaIntegrator.h = par.dt;
	tea.Ntot = par.Ntot;
	 
	hipMalloc(&tea.rforce, par.Ntr * par.Ntot * sizeof(float4));
	hipMalloc(&tea.mforce, par.Ntr * par.Ntot * sizeof(float4));
	hipMalloc(&tea.coords, par.Ntr * par.Ntot * sizeof(float4));

	hipMalloc(&tea.d_epsilon, par.Ntot * par.Ntr * sizeof(float));
	hipMalloc(&tea.d_ci, par.Ntot * par.Ntr * sizeof(float4));
	hipMalloc(&tea.d_beta_ij, par.Ntr * sizeof(float));
    tea.h_epsilon = (float*) malloc(par.Ntot * par.Ntr * sizeof(float));
	tea.h_beta_ij = (float*) malloc( par.Ntr * sizeof(float));

	checkCUDAError("tea memory allocation");
	hipMemcpyToSymbol(HIP_SYMBOL(c_tea), &tea, sizeof(Tea), 0, hipMemcpyHostToDevice);
	checkCUDAError("tea constant copy");

    //createTeaUpdater();
    printf("TEA integrator initialized, a = %f A, freq = %d steps; capricious mode: %s; using pairlist: %s; block size: %d\n", tea.a, tea.epsilon_freq, (tea.capricious ? "on" : "off"), (tea.unlisted ? "no" : "yes"), BLOCK_SIZE);   
}

void integrateTea(){
	// Pregenerate random forces
	integrateTea_prepare<<<(par. Ntr * par.Ntot) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_f, d_r);
	integrateTea_kernel_unlisted<<<(par. Ntr * par.Ntot) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_f, d_r);
	checkCUDAError("integrate TEA prepare and kernel");
}

void deleteTeaIntegrator(){
	//deleteLangevinIntegrator();
	hipFree(tea.rforce);
	hipFree(tea.mforce);
	hipFree(tea.coords);
   	hipFree(tea.d_epsilon);
	hipFree(tea.d_beta_ij);
	hipFree(tea.d_ci);
    free(tea.h_epsilon);
	free(tea.h_beta_ij);

}

void updateTea(long long int step){
	const int update_epsilon = (step % tea.epsilon_freq) == 0;
	const int N = par.Ntot;

	int* Nnoextra = (int*)calloc(par.Ntr, sizeof(int));
	for(int tr = 0; tr < par.Ntr; tr++){
		for(int j = 0; j < par.Ntot; j++){
			if (!top.extra[j + par.Ntot * tr]){
				Nnoextra[tr]++;
			}
		}
		//printf("Nnoextra for [%d] is %d\n", tr, Nnoextra[tr]);
	}

	if (update_epsilon){
		// Calculate relative coupling
		integrateTea_epsilon_unlisted<<<(par.Ntr * par.Ntot) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_r);
		// Dowload epsilon`s
		hipMemcpy(tea.h_epsilon, tea.d_epsilon, par.Ntr * par.Ntot * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAError("copy epsilon from device");

		//printf("epsilon: [ ");
		for (int t = 0; t < par.Ntr; ++t){
			double epsilon = 0.0;

			for (int i = 0; i < N; ++i){
				//if (i%10==0) printf("[%d] %f\n", i, tea.h_epsilon[t*N + i]);
				epsilon += tea.h_epsilon[t*N + i];
			}

			//epsilon /= 3.*N*(3.*N - 3.); // Averaging, off-diagonal elements only
			epsilon /= 3.*Nnoextra[t]*(3.*Nnoextra[t] - 3.);
			if (epsilon > 1.0){
				if (tea.capricious){
					printf("HI tensor is not diagonal enough for trajectory %d: epsilon = %lf -> 1.0!\n", t, epsilon);
					exit(-1);
				}
				epsilon = 1.0;
			}
			if (epsilon > tea.epsmax){
				printf("HI tensor is not diagonal enough for trajectory %d: epsilon = %lf > %f = tea_epsmax!\n", t, epsilon, tea.epsmax);
				exit(-1);
			}
			//double a = (3.*N-1.)*epsilon*epsilon - (3.*N-2.)*epsilon;
			double a = (3.*Nnoextra[t]-1.)*epsilon*epsilon - (3.*Nnoextra[t]-2.)*epsilon;
			if (fabs(a) < 1e-7){ // To avoid 0/0 division in eq. (26) we explicitly handle small a's
				tea.h_beta_ij[t] = .5f;
				if(tea.capricious && tea.a > 0.0f){
					printf("HI tensor is too diagonal for trajectory %d: a = %lf, beta: %lf -> 0.5!\n", t, a, (1. - sqrt(1. - a)) / a);
					exit(-1);
				}
			} else {
				tea.h_beta_ij[t] = (1. - sqrt(1. - a)) / a; // eq. (26)
			}
			tea.h_epsilon[t] = epsilon; // We slowly overwrite the beginning of h_epsilon with per-trajectory epsilons to later output them
			//printf("%lf ", epsilon);
		}
		//printf("]\n");
		hipMemcpy(tea.d_beta_ij, tea.h_beta_ij, par.Ntr * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("copy betaij to device");
	}
}


